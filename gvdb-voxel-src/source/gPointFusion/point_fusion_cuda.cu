#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------
//
// FLUIDS v.3 - SPH Fluid Simulator for CPU and GPU
// Copyright (C) 2012-2013. Rama Hoetzlein, http://fluids3.com
//
// BSD 3-clause:
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this 
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this 
//    list of conditions and the following disclaimer in the documentation and/or 
//    other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may 
//    be used to endorse or promote products derived from this software without specific 
//   prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT 
// OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) 
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
// TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
// EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//----------------------------------------------------------------------------------

#define CUDA_KERNEL
#include "point_fusion_cuda.cuh"
#include "cutil_math.h"			// cutil32.lib
#include <string.h>
#include <assert.h>

struct ALIGN(16) Obj {
	float3		pos;
	float3		size;
	float3		loc;
	uint		clr;
};

struct ALIGN(16) ScanInfo {
	int*		objGrid;
	int*		objCnts;
	Obj*		objList;
	float3*		pntList;
	uint*		pntClrs;
	int3		gridRes;
	float3		gridSize;	
	float3		cams;
	float3		camu;
	float3		camv;
	uint*		rnd_seeds;
};
__device__ ScanInfo		scan;
__device__ int			pntout;

// Generate random unsigned int in [0, 2^24)
static __host__ __device__ __inline__ unsigned int lcg(unsigned int &prev)
{
  const unsigned int LCG_A = 1664525u;
  const unsigned int LCG_C = 1013904223u;
  prev = (LCG_A * prev + LCG_C);
  return prev & 0x00FFFFFF;
}
static __host__ __device__ __inline__ float rnd(unsigned int &prev)
{
  return ((float) lcg(prev) / (float) 0x01000000);
}

// Get view ray
inline __device__ float3 getViewRay ( float x, float y )
{
	float3 v = x*scan.camu + y*scan.camv + scan.cams;  
	return normalize(v);
}

#define NOHIT			1.0e10f

// Ray box intersection
inline __device__ float3 rayBoxIntersect ( float3 rpos, float3 rdir, float3 vmin, float3 vmax )
{
	register float ht[8];
	ht[0] = (vmin.x - rpos.x)/rdir.x;
	ht[1] = (vmax.x - rpos.x)/rdir.x;
	ht[2] = (vmin.y - rpos.y)/rdir.y;
	ht[3] = (vmax.y - rpos.y)/rdir.y;
	ht[4] = (vmin.z - rpos.z)/rdir.z;
	ht[5] = (vmax.z - rpos.z)/rdir.z;
	ht[6] = fmax(fmax(fmin(ht[0], ht[1]), fmin(ht[2], ht[3])), fmin(ht[4], ht[5]));
	ht[7] = fmin(fmin(fmax(ht[0], ht[1]), fmax(ht[2], ht[3])), fmax(ht[4], ht[5]));	
	ht[6] = (ht[6] < 0 ) ? 0.0 : ht[6];
	return make_float3( ht[6], ht[7], (ht[7]<ht[6] || ht[7]<0) ? NOHIT : 0 );
}

#define COLOR(r,g,b)	( (uint((b)*255.0f)<<16) | (uint((g)*255.0f)<<8) | uint((r)*255.0f) ) 

float3 __device__ __inline__ jitter_sample ()
{	 
	uint index = (threadIdx.y % 128) * 128 + (threadIdx.x % 128);
    unsigned int seed  = scan.rnd_seeds[ index ]; 
    float uu = rnd( seed );
    float vv = rnd( seed );
	float ww = rnd( seed );   
	scan.rnd_seeds[ index ] = seed;
    return make_float3(uu,vv,ww);
}

extern "C" __global__ void scanBuildings ( float3 pos, int3 res, int num_obj, float tmax )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ( x >= res.x || y >= res.y ) return;

	float3 jit = jitter_sample();
	float3 dir = getViewRay( float(x+jit.x)/float(res.x), float(y+jit.y)/float(res.y) );
	
	int gcell = int(pos.z/scan.gridSize.y) * scan.gridRes.x + int(pos.x/scan.gridSize.x);
	if ( gcell < 0 || gcell > scan.gridRes.x*scan.gridRes.y)  return;

	Obj* bldg;
	float3 t, tnearest;
	uint clr = 0;

	tnearest.x = NOHIT;	

	//for (int n=0; n < scan.objCnts[gcell]; n++) {
//		bldg = scan.objList + (scan.objGrid[gcell] + n);

	for (int n=0; n < num_obj; n++) {
		bldg = scan.objList + n;
		if ( bldg != 0 ) {
			t = rayBoxIntersect ( pos, dir, bldg->pos, bldg->pos + bldg->size );
			if ( t.x < tnearest.x && t.x < tmax && t.z != NOHIT ) {
				tnearest = t;
				clr = bldg->clr;
			}
		}
	}
	if ( tnearest.x == NOHIT) { scan.pntList[ y*res.x + x] = make_float3(0,0,0); return; }

	atomicAdd(&pntout, 1);
	
	scan.pntList[ y*res.x + x] = pos + tnearest.x * dir;
	scan.pntClrs[ y*res.x + x] = clr;	
}



