#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------
//
// FLUIDS v.3 - SPH Fluid Simulator for CPU and GPU
// Copyright (C) 2012-2013. Rama Hoetzlein, http://fluids3.com
//
// BSD 3-clause:
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this 
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this 
//    list of conditions and the following disclaimer in the documentation and/or 
//    other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may 
//    be used to endorse or promote products derived from this software without specific 
//   prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT 
// OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) 
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
// TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
// EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//----------------------------------------------------------------------------------

#define CUDA_KERNEL
#include "fluid_system_cuda.cuh"

#include "cutil_math.h"			// cutil32.lib
#include <string.h>
#include <assert.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__constant__ FParams		fparam;			// CPU Fluid params
__constant__ FBufs			fbuf;			// GPU Particle buffers (unsorted)
__constant__ FBufs			ftemp;			// GPU Particle buffers (sorted)
__constant__ uint			gridActive;

#define SCAN_BLOCKSIZE		512

extern "C" __global__ void insertParticles ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	//-- debugging (pointers should match CUdeviceptrs on host side)
	// printf ( " pos: %012llx, gcell: %012llx, gndx: %012llx, gridcnt: %012llx\n", fbuf.bufC(FPOS), fbuf.bufC(FGCELL), fbuf.bufC(FGNDX), fbuf.bufC(FGRIDCNT) );

	register float3 gridMin =	fparam.gridMin;
	register float3 gridDelta = fparam.gridDelta;
	register int3 gridRes =		fparam.gridRes;
	register int3 gridScan =	fparam.gridScanMax;

	register int		gs;
	register float3		gcf;
	register int3		gc;	

	gcf = (fbuf.bufF3(FPOS)[i] - gridMin) * gridDelta; 
	gc = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) );
	gs = (gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;

	if ( gc.x >= 1 && gc.x <= gridScan.x && gc.y >= 1 && gc.y <= gridScan.y && gc.z >= 1 && gc.z <= gridScan.z ) {
		fbuf.bufI(FGCELL)[i] = gs;											// Grid cell insert.
		fbuf.bufI(FGNDX)[i] = atomicAdd ( &fbuf.bufI(FGRIDCNT)[ gs ], 1 );		// Grid counts.

		//gcf = (-make_float3(poff,poff,poff) + fbuf.bufF3(FPOS)[i] - gridMin) * gridDelta;
		//gc = make_int3( int(gcf.x), int(gcf.y), int(gcf.z) );
		//gs = ( gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;
	} else {
		fbuf.bufI(FGCELL)[i] = GRID_UNDEF;		
	}
}

// Counting Sort - Full (deep copy)
extern "C" __global__ void countingSortFull ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;		// particle index				
	if ( i >= pnum ) return;

	// Copy particle from original, unsorted buffer (msortbuf),
	// into sorted memory location on device (mpos/mvel)
	uint icell = ftemp.bufI(FGCELL) [ i ];	

	if ( icell != GRID_UNDEF ) {	  
		// Determine the sort_ndx, location of the particle after sort		
		uint indx =  ftemp.bufI(FGNDX)  [ i ];		
	    int sort_ndx = fbuf.bufI(FGRIDOFF) [ icell ] + indx ;	// global_ndx = grid_cell_offet + particle_offset	
		//printf ( "%d: cell: %d, off: %d, ndx: %d\n", i, icell, fbuf.bufI(FGRIDOFF)[icell], indx );
		
		// Transfer data to sort location
		fbuf.bufI (FGRID) [ sort_ndx ] =	sort_ndx;			// full sort, grid indexing becomes identity		
		fbuf.bufF3(FPOS) [sort_ndx] =		ftemp.bufF3(FPOS) [i];
		fbuf.bufF3(FVEL) [sort_ndx] =		ftemp.bufF3(FVEL) [i];
		fbuf.bufF3(FVEVAL)[sort_ndx] =		ftemp.bufF3(FVEVAL) [i];
		fbuf.bufF3(FFORCE)[sort_ndx] =		ftemp.bufF3(FFORCE) [i];
		fbuf.bufF (FPRESS)[sort_ndx] =		ftemp.bufF(FPRESS) [i];
		fbuf.bufF (FDENSITY)[sort_ndx] =	ftemp.bufF(FDENSITY) [i];
		fbuf.bufI (FCLR) [sort_ndx] =		ftemp.bufI(FCLR) [i];
		fbuf.bufI (FGCELL) [sort_ndx] =		icell;
		fbuf.bufI (FGNDX) [sort_ndx] =		indx;		
	}
} 

extern "C" __device__ float contributePressure ( int i, float3 p, int cell )
{			
	if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return 0.0;

	float3 dist;
	float dsq, c, sum = 0.0;
	register float d2 = fparam.psimscale * fparam.psimscale;
	register float r2 = fparam.r2 / d2;
	
	int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];

	for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {
		int pndx = fbuf.bufI(FGRID) [cndx];
		dist = p - fbuf.bufF3(FPOS) [pndx];
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		if ( dsq < r2 && dsq > 0.0) {
			c = (r2 - dsq)*d2;
			sum += c * c * c;				
		} 
	}
	
	return sum;
}
			
extern "C" __global__ void computePressure ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint gc = fbuf.bufI(FGCELL) [i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= nadj;

	// Sum Pressures
	float3 pos = fbuf.bufF3(FPOS) [i];
	float sum = 0.0;
	for (int c=0; c < fparam.gridAdjCnt; c++) {
		sum += contributePressure ( i, pos, gc + fparam.gridAdj[c] );
	}
	__syncthreads();
		
	// Compute Density & Pressure
	sum = sum * fparam.pmass * fparam.poly6kern;
	if ( sum == 0.0 ) sum = 1.0;
	fbuf.bufF(FPRESS)  [ i ] = ( sum - fparam.prest_dens ) * fparam.pintstiff;
	fbuf.bufF(FDENSITY)[ i ] = 1.0f / sum;
}

extern "C" __device__ float3 contributeForce ( int i, float3 ipos, float3 iveleval, float ipress, float idens, int cell)
{			
	if ( fbuf.bufI(FGRIDCNT)[cell] == 0 ) return make_float3(0,0,0);	

	float dsq, c, pterm;	
	float3 dist, force = make_float3(0,0,0);
	int j;

	int clast = fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell];

	for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < clast; cndx++ ) {
		j = fbuf.bufI(FGRID)[ cndx ];				
		dist = ( ipos - fbuf.bufF3(FPOS)[ j ] );		// dist in cm
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		if ( dsq < fparam.rd2 && dsq > 0) {			
			dsq = sqrt(dsq * fparam.d2);
			c = ( fparam.psmoothradius - dsq ); 
			pterm = fparam.psimscale * -0.5f * c * fparam.spikykern * ( ipress + fbuf.bufF(FPRESS)[ j ] ) / dsq;			
			force += ( pterm * dist + fparam.vterm * ( fbuf.bufF3(FVEVAL)[ j ] - iveleval )) * c * idens * (fbuf.bufF(FDENSITY)[ j ] );
		}	
	}
	return force;
}


extern "C" __global__ void computeForce ( int pnum)
{			
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell	
	uint gc = fbuf.bufI(FGCELL)[ i ];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;

	// Sum Pressures	
	register float3 force;
	force = make_float3(0,0,0);		

	for (int c=0; c < fparam.gridAdjCnt; c++) {
		force += contributeForce ( i, fbuf.bufF3(FPOS)[ i ], fbuf.bufF3(FVEVAL)[ i ], fbuf.bufF(FPRESS)[ i ], fbuf.bufF(FDENSITY)[ i ], gc + fparam.gridAdj[c] );
	}
	fbuf.bufF3(FFORCE)[ i ] = force;
}

extern "C" __global__ void randomInit ( int seed, int numPnts )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts ) return;

	// Initialize particle random generator	
	hiprandState_t* st = (hiprandState_t*) (fbuf.bufC(FSTATE) + i*sizeof(hiprandState_t));
	hiprand_init ( seed + i, 0, 0, st );		
}

#define CURANDMAX		2147483647

extern "C" __global__ void emitParticles ( float frame, int emit, int numPnts )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= emit ) return;

	hiprandState_t* st = (hiprandState_t*) (fbuf.bufC(FSTATE) + i*sizeof(hiprandState_t));
	uint v = hiprand( st);
	uint j = v & (numPnts-1);
	float3 bmin = make_float3(-170,10,-20);
	float3 bmax = make_float3(-190,60, 20);

	float3 pos = make_float3(0,0,0);	
	pos.x = float( v & 0xFF ) / 256.0;
	pos.y = float((v>>8) & 0xFF ) / 256.0;
	pos.z = float((v>>16) & 0xFF ) / 256.0;
	pos = bmin + pos*(bmax-bmin);	
	
	fbuf.bufF3(FPOS)[j] = pos;
	fbuf.bufF3(FVEVAL)[j] = make_float3(0,0,0);
	fbuf.bufF3(FVEL)[j] = make_float3(5,-2,0);
	fbuf.bufF3(FFORCE)[j] = make_float3(0,0,0);	
	
}

__device__ uint getGridCell ( float3 pos, uint3& gc )
{	
	gc.x = (int)( (pos.x - fparam.gridMin.x) * fparam.gridDelta.x);			// Cell in which particle is located
	gc.y = (int)( (pos.y - fparam.gridMin.y) * fparam.gridDelta.y);
	gc.z = (int)( (pos.z - fparam.gridMin.z) * fparam.gridDelta.z);		
	return (int) ( (gc.y*fparam.gridRes.z + gc.z)*fparam.gridRes.x + gc.x);	
}

extern "C" __global__ void sampleParticles ( float* brick, uint3 res, float3 bmin, float3 bmax, int numPnts, float scalar )
{
	float3 dist;
	float dsq;
	int j, cell;	
	register float r2 = fparam.r2;
	register float h2 = 2.0*r2 / 8.0;		// 8.0=smoothing. higher values are sharper

	uint3 i = blockIdx * make_uint3(blockDim.x, blockDim.y, blockDim.z) + threadIdx;
	if ( i.x >= res.x || i.y >= res.y || i.z >= res.z ) return;
	
	float3 p = bmin + make_float3(float(i.x)/res.x, float(i.y)/res.y, float(i.z)/res.z) * (bmax-bmin);
	//float3 v = make_float3(0,0,0);
	float v = 0.0;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint3 gc;
	uint gs = getGridCell ( p, gc );
	if ( gc.x < 1 || gc.x > fparam.gridRes.x-fparam.gridSrch || gc.y < 1 || gc.y > fparam.gridRes.y-fparam.gridSrch || gc.z < 1 || gc.z > fparam.gridRes.z-fparam.gridSrch ) {
		brick[ (i.y*int(res.z) + i.z)*int(res.x) + i.x ] = 0.0;
		return;
	}

	gs -= nadj;	

	for (int c=0; c < fparam.gridAdjCnt; c++) {
		cell = gs + fparam.gridAdj[c];		
		if ( fbuf.bufI(FGRIDCNT)[cell] != 0 ) {				
			for ( int cndx = fbuf.bufI(FGRIDOFF)[cell]; cndx < fbuf.bufI(FGRIDOFF)[cell] + fbuf.bufI(FGRIDCNT)[cell]; cndx++ ) {
				j = fbuf.bufI(FGRID)[cndx];
				dist = p - fbuf.bufF3(FPOS)[ j ];
				dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
				if ( dsq < fparam.rd2 && dsq > 0 ) {
					dsq = sqrt(dsq * fparam.d2);					
					//v += fbuf.mvel[j] * (fparam.gausskern * exp ( -(dsq*dsq)/h2 ) / fbuf.mdensity[ j ]);
					v += fparam.gausskern * exp ( -(dsq*dsq)/h2 );
				}
			}
		}
	}
	__syncthreads();

	brick[ (i.z*int(res.y) + i.y)*int(res.x) + i.x ] = v * scalar;
	//brick[ (i.z*int(res.y) + i.y)*int(res.x) + i.x ] = length(v) * scalar;
}

extern "C" __global__ void computeQuery ( int pnum )
{
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	// Get search cell
	int nadj = (1*fparam.gridRes.z + 1)*fparam.gridRes.x + 1;
	uint gc = fbuf.bufI(FGCELL) [i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	gc -= nadj;

	// Sum Pressures
	float sum = 0.0;
	for (int c=0; c < fparam.gridAdjCnt; c++) {
		sum += 1.0;
	}
	__syncthreads();
	
}

		
extern "C" __global__ void advanceParticles ( float time, float dt, float ss, int numPnts )
{		
	uint i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;	// particle index				
	if ( i >= numPnts ) return;
	
	if ( fbuf.bufI(FGCELL)[i] == GRID_UNDEF ) {
		fbuf.bufF3(FPOS)[i] = make_float3(-1000,-1000,-1000);
		fbuf.bufF3(FVEL)[i] = make_float3(0,0,0);
		return;
	}
			
	// Get particle vars
	register float3 accel, norm;
	register float diff, adj, speed;
	register float3 pos = fbuf.bufF3(FPOS)[i];
	register float3 veval = fbuf.bufF3(FVEVAL)[i];

	// Leapfrog integration						
	accel = fbuf.bufF3(FFORCE)[i];
	accel *= fparam.pmass;	
		
	// Boundaries
	// Y-axis
	
	diff = fparam.pradius - (pos.y - (fparam.pboundmin.y + (pos.x-fparam.pboundmin.x)*fparam.pground_slope )) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( -fparam.pground_slope, 1.0 - fparam.pground_slope, 0);
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	diff = fparam.pradius - ( fparam.pboundmax.y - pos.y )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(0, -1, 0);
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// X-axis
	diff = fparam.pradius - (pos.x - (fparam.pboundmin.x + (sin(time*fparam.pforce_freq)+1)*0.5 * fparam.pforce_min))*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 1, 0, 0);
		adj = (fparam.pforce_min+1) * fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = fparam.pradius - ( (fparam.pboundmax.x - (sin(time*fparam.pforce_freq)+1)*0.5*fparam.pforce_max) - pos.x)*ss;
	if ( diff > EPSILON ) {
		norm = make_float3(-1, 0, 0);
		adj = (fparam.pforce_max+1) * fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}

	// Z-axis
	diff = fparam.pradius - (pos.z - fparam.pboundmin.z ) * ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, 1 );
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
	diff = fparam.pradius - ( fparam.pboundmax.z - pos.z )*ss;
	if ( diff > EPSILON ) {
		norm = make_float3( 0, 0, -1 );
		adj = fparam.pextstiff * diff - fparam.pdamp * dot(norm, veval );
		norm *= adj; accel += norm;
	}
		
	// Gravity
	accel += fparam.pgravity;

	// Accel Limit
	speed = accel.x*accel.x + accel.y*accel.y + accel.z*accel.z;
	if ( speed > fparam.AL2 ) {
		accel *= fparam.AL / sqrt(speed);
	}

	// Velocity Limit
	float3 vel = fbuf.bufF3(FVEL)[i];
	speed = vel.x*vel.x + vel.y*vel.y + vel.z*vel.z;
	if ( speed > fparam.VL2 ) {
		speed = fparam.VL2;
		vel *= fparam.VL / sqrt(speed);
	}

	// Ocean colors
	/*uint clr = fbuf.bufI(FCLR)[i];
	if ( speed > fparam.VL2*0.2) {
		adj = fparam.VL2*0.2;		
		clr += ((  clr & 0xFF) < 0xFD ) ? +0x00000002 : 0;		// decrement R by one
		clr += (( (clr>>8) & 0xFF) < 0xFD ) ? +0x00000200 : 0;	// decrement G by one
		clr += (( (clr>>16) & 0xFF) < 0xFD ) ? +0x00020000 : 0;	// decrement G by one
		fbuf.bufI(FCLR)[i] = clr;
	}
	if ( speed < 0.03 ) {		
		int v = int(speed/.01)+1;
		clr += ((  clr & 0xFF) > 0x80 ) ? -0x00000001 * v : 0;		// decrement R by one
		clr += (( (clr>>8) & 0xFF) > 0x80 ) ? -0x00000100 * v : 0;	// decrement G by one
		fbuf.bufI(FCLR)[i] = clr;
	}*/
	
	//-- surface particle density 
	//fbuf.mclr[i] = fbuf.mclr[i] & 0x00FFFFFF;
	//if ( fbuf.mdensity[i] > 0.0014 ) fbuf.mclr[i] += 0xAA000000;

	// Leap-frog Integration
	float3 vnext = accel*dt + vel;					// v(t+1/2) = v(t-1/2) + a(t) dt		
	fbuf.bufF3(FVEVAL)[i] = (vel + vnext) * 0.5;	// v(t+1) = [v(t-1/2) + v(t+1/2)] * 0.5			
	fbuf.bufF3(FVEL)[i] = vnext;
	fbuf.bufF3(FPOS)[i] += vnext * (dt/ss);			// p(t+1) = p(t) + v(t+1/2) dt		
}


extern "C" __global__ void prefixFixup(uint *input, uint *aux, int len)
{
	unsigned int t = threadIdx.x;
	unsigned int start = t + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	if (start < len)					input[start] += aux[blockIdx.x];
	if (start + SCAN_BLOCKSIZE < len)   input[start + SCAN_BLOCKSIZE] += aux[blockIdx.x];
}

extern "C" __global__ void prefixSum(uint* input, uint* output, uint* aux, int len, int zeroff)
{
	__shared__ uint scan_array[SCAN_BLOCKSIZE << 1];
	unsigned int t1 = threadIdx.x + 2 * blockIdx.x * SCAN_BLOCKSIZE;
	unsigned int t2 = t1 + SCAN_BLOCKSIZE;

	// Pre-load into shared memory
	scan_array[threadIdx.x] = (t1<len) ? input[t1] : 0.0f;
	scan_array[threadIdx.x + SCAN_BLOCKSIZE] = (t2<len) ? input[t2] : 0.0f;
	__syncthreads();

	// Reduction
	int stride;
	for (stride = 1; stride <= SCAN_BLOCKSIZE; stride <<= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index < 2 * SCAN_BLOCKSIZE)
			scan_array[index] += scan_array[index - stride];
		__syncthreads();
	}

	// Post reduction
	for (stride = SCAN_BLOCKSIZE >> 1; stride > 0; stride >>= 1) {
		int index = (threadIdx.x + 1) * stride * 2 - 1;
		if (index + stride < 2 * SCAN_BLOCKSIZE)
			scan_array[index + stride] += scan_array[index];
		__syncthreads();
	}
	__syncthreads();

	// Output values & aux
	if (t1 + zeroff < len)	output[t1 + zeroff] = scan_array[threadIdx.x];
	if (t2 + zeroff < len)	output[t2 + zeroff] = (threadIdx.x == SCAN_BLOCKSIZE - 1 && zeroff) ? 0 : scan_array[threadIdx.x + SCAN_BLOCKSIZE];
	if (threadIdx.x == 0) {
		if (zeroff) output[0] = 0;
		if (aux) aux[blockIdx.x] = scan_array[2 * SCAN_BLOCKSIZE - 1];
	}
}

